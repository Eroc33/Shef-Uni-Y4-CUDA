#include "hip/hip_runtime.h"
#include "../rgb.h"
#include <omp.h>
#include <math.h>
#include <stdio.h>

__global__ void kernel(rgb* data, big_rgb* work_buffer, unsigned int width, unsigned int height, unsigned int wb_width, unsigned wb_height, unsigned int c){

    unsigned int x = (blockIdx.x * 32) + threadIdx.x;
    unsigned int y = (blockIdx.y * 32) + threadIdx.y;

	if (x >= wb_width || y >= wb_height) {
		//this is not at all efficient :(
		return;
	}
    
    //cell height is input cell size
    unsigned int cell_height = c;
    //unless this is the last row of cells, and c does not evenly divide height
    if (height%cell_height != 0 && y + 1 == wb_height) {
        cell_height = height % cell_height;
    }
    unsigned int cell_y = y*c;
    //cell width is input cell size
    unsigned int cell_width = c;
    //unless this is the last column of cells, and c does not evenly divide width
    if (width%cell_width != 0 && x + 1 == wb_width) {
        cell_width = width % cell_width;
    }
    unsigned int cell_size = cell_width * cell_height;
    big_rgb avg = { 0,0,0 };

    unsigned int cell_x = x*c;

    //averages
    for (unsigned int cy = 0; cy < cell_height; cy++) {
        for (unsigned int cx = 0; cx < cell_width; cx++) {
            int i = ((cell_y + cy)*height) + cell_x + cx;
            //partial global sum
            rgb_generic_add_assign(&work_buffer[y*wb_width+x], &data[i]);
            //cell sum
            rgb_generic_add_assign(&avg, &data[i]);
        }
    }

    rgb_generic_div_assign(&avg, cell_size);
    rgb out = {
        (unsigned char)avg.r,
        (unsigned char)avg.g,
        (unsigned char)avg.b
    };

    //copy to all ouput buffer cells
    for (unsigned int cy = 0; cy < cell_height; cy++) {
        for (unsigned int cx = 0; cx < cell_width; cx++) {
            int i = ((cell_y + cy)*height) + cell_x + cx;
            data[i] = out;
        }
    }
}

#define bail_on_cuda_error(err) _bail_on_cuda_error(err,__FILE__,__LINE__)

void _bail_on_cuda_error(hipError_t err, const char* file, unsigned int line){
    if(err == hipSuccess){
        return;
    }
    fprintf(stderr,"FATAL: Encountered cuda error: %s (%s) (%s,%u).\n Quitting.\n",hipGetErrorName(err),hipGetErrorString(err),file,line);
    exit(EXIT_FAILURE);
}

void run_cuda(big_rgb* work_buffer, rgb* data, unsigned int width, unsigned int height, unsigned int wb_width, unsigned int wb_height, unsigned int c){
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	//starting cpu based timing here
	double begin = omp_get_wtime();

    big_rgb global_avg = {0,0,0};
    
    rgb* gpu_data;
    big_rgb* gpu_wb;

    bail_on_cuda_error(hipMalloc((void**)&gpu_data,width*height*sizeof(rgb)));
    bail_on_cuda_error(hipMalloc((void**)&gpu_wb,wb_width*wb_height*sizeof(big_rgb)));

    bail_on_cuda_error(hipMemcpy(gpu_data, data, width*height*sizeof(rgb), hipMemcpyHostToDevice));
    bail_on_cuda_error(hipMemset(gpu_wb, 0, wb_width*wb_height*sizeof(rgb)));

    //run kernel code
    dim3 blocksPerGrid((wb_width+(32 - 1))/32,(wb_height+(32 - 1))/32,1);
    dim3 threadsPerBlock(32,32,1);

	hipEventRecord(start);
    kernel<<<blocksPerGrid, threadsPerBlock>>>(gpu_data,gpu_wb,width,height,wb_width,wb_height,c);
	hipEventRecord(stop);
	hipEventSynchronize(stop);
    bail_on_cuda_error(hipGetLastError());

    bail_on_cuda_error(hipMemcpy(work_buffer, gpu_wb, wb_width*wb_height*sizeof(big_rgb), hipMemcpyDeviceToHost));

    //TODO: use a cuda reduction to speed this up & avoid copying wb to cpu
    for(int i=0;i<wb_width*wb_height;i++){
		global_avg.r += work_buffer[i].r;
		global_avg.g += work_buffer[i].g;
		global_avg.b += work_buffer[i].b;
	}
	//divide by count for global average
    big_rgb_div_assign(&global_avg, width*height);

    bail_on_cuda_error(hipMemcpy(data, gpu_data, width*height*sizeof(rgb), hipMemcpyDeviceToHost));
    
    bail_on_cuda_error(hipFree(gpu_wb));
    bail_on_cuda_error(hipFree(gpu_data));

	// Output the average colour value for the image
	printf("CUDA Average image colour red = %u, green = %u, blue = %u \n",(unsigned char)global_avg.r,(unsigned char)global_avg.g,(unsigned char)global_avg.b);

	//end timing here
	double end = omp_get_wtime();
	double seconds = (end - begin);

	float cudaMs;

	hipEventElapsedTime(&cudaMs,start,stop);

	double s;
	double ms = modf(seconds,&s)*1000.0;
	printf("CUDA mode execution time took %d s and %dms (%f ms as measured by cuda)\n",(int)s,(int)ms,cudaMs);
}