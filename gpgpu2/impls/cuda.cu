#include "hip/hip_runtime.h"
#include "../rgb.h"
#include <omp.h>
#include <math.h>
#include <stdio.h>

#define cuda_check_error(err_expr)\
{\
	hipError_t err = (err_expr);\
	if(err != hipSuccess){\
		printf("FATAL: Encountered cuda error: %s (%s) (%s,%u).\n Quitting.\n", hipGetErrorName(err), hipGetErrorString(err), __FILE__, __LINE__);\
		assert(0);\
	}\
}

//requirements:
//   blockDim.x == c, blockDim.y == 1
//   gridDim.x == num_cells_x , gridDim.y == 1
__global__ void row_reduction(rgb* data, unsigned int width, unsigned int height) {
	extern __shared__ unsigned char sdata[];

	unsigned char* r = &sdata[0 * blockDim.x];
	unsigned char* g = &sdata[1 * blockDim.x];
	unsigned char* b = &sdata[2 * blockDim.x];

	unsigned int y = blockIdx.y;
	unsigned int cell_start = blockIdx.x*blockDim.x;
	unsigned int px_x = cell_start + threadIdx.x;
	unsigned int y_offset = (y*width);

	if (px_x < width) {
		r[threadIdx.x] = data[px_x + y_offset].r;
		g[threadIdx.x] = data[px_x + y_offset].g;
		b[threadIdx.x] = data[px_x + y_offset].b;
	}
	__syncthreads();
	for (unsigned int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
		if (threadIdx.x < stride && px_x+stride < width) {
			r[threadIdx.x] = ((unsigned int)r[threadIdx.x] + (unsigned int)r[threadIdx.x + stride]) / 2;
			g[threadIdx.x] = ((unsigned int)g[threadIdx.x] + (unsigned int)g[threadIdx.x + stride]) / 2;
			b[threadIdx.x] = ((unsigned int)b[threadIdx.x] + (unsigned int)b[threadIdx.x + stride]) / 2;
		}
		__syncthreads();
	}

	if (threadIdx.x == 0) {
		data[px_x + y_offset].r = r[threadIdx.x];
		data[px_x + y_offset].g = g[threadIdx.x];
		data[px_x + y_offset].b = b[threadIdx.x];
	}
}

//col_reduction, similar to row_reduction but reduce the already reduced rows into columns
__global__ void col_reduction(rgb* data, unsigned int width, unsigned int height, unsigned int c) {
	extern __shared__ unsigned char sdata[];

	unsigned char* r = &sdata[0 * blockDim.x];
	unsigned char* g = &sdata[1 * blockDim.x];
	unsigned char* b = &sdata[2 * blockDim.x];

	//load sdata
	unsigned int x = blockIdx.y*c;
	unsigned int cell_start_y = blockIdx.x*blockDim.x;
	unsigned int px_y = cell_start_y + threadIdx.x;
	unsigned int px_pos = x+(px_y*width);
	if (px_y < height) {
		r[threadIdx.x] = data[px_pos].r;
		g[threadIdx.x] = data[px_pos].g;
		b[threadIdx.x] = data[px_pos].b;
	}
	__syncthreads();
	for (unsigned int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
		if (threadIdx.x < stride && px_y+stride < height) {
			r[threadIdx.x] = ((unsigned int)r[threadIdx.x] + (unsigned int)r[threadIdx.x + stride]) / 2;
			g[threadIdx.x] = ((unsigned int)g[threadIdx.x] + (unsigned int)g[threadIdx.x + stride]) / 2;
			b[threadIdx.x] = ((unsigned int)b[threadIdx.x] + (unsigned int)b[threadIdx.x + stride]) / 2;
		}
		__syncthreads();
	}

	if (threadIdx.x == 0) {
		data[px_pos].r = r[threadIdx.x];
		data[px_pos].g = g[threadIdx.x];
		data[px_pos].b = b[threadIdx.x];
	}
}

__global__ void scatter(rgb* data, unsigned int width, unsigned int height, unsigned int wb_width, unsigned wb_height, unsigned int c){

    unsigned int x = (blockIdx.x * 32) + threadIdx.x;
    unsigned int y = (blockIdx.y * 32) + threadIdx.y;

	if (x >= wb_width || y >= wb_height) {
		//this is not at all efficient :(
		return;
	}
    
    //cell height is input cell size
    unsigned int cell_height = c;
    //unless this is the last row of cells, and c does not evenly divide height
    if (height%cell_height != 0 && y + 1 == wb_height) {
        cell_height = height % cell_height;
    }
    unsigned int cell_y = y*c;
    //cell width is input cell size
    unsigned int cell_width = c;
    //unless this is the last column of cells, and c does not evenly divide width
    if (width%cell_width != 0 && x + 1 == wb_width) {
        cell_width = width % cell_width;
    }
    unsigned int cell_size = cell_width * cell_height;
    unsigned int cell_x = x*c;

	rgb out = data[cell_y*width + cell_x];

    //copy to all ouput buffer cells
    for (unsigned int cy = cell_y; cy < cell_y+cell_height; cy++) {
        for (unsigned int cx = cell_x; cx < cell_x+cell_width; cx++) {
            data[(cy*width) + cx] = out;
        }
    }
}

void run_cuda(big_rgb* work_buffer, rgb* data, unsigned int width, unsigned int height, unsigned int wb_width, unsigned int wb_height, unsigned int c){
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	//starting cpu based timing here
	double begin = omp_get_wtime();

	int num_cells_x = (width + (c - 1)) / c;
	int num_cells_y = (height + (c - 1)) / c;

    big_rgb global_avg = {0,0,0};
    
    rgb* gpu_data;

    cuda_check_error(hipMalloc((void**)&gpu_data,width*height*sizeof(rgb)));
    cuda_check_error(hipMemcpy(gpu_data, data, width*height*sizeof(rgb), hipMemcpyHostToDevice));

    //run kernel code
    dim3 blocksPerGrid((wb_width+(32 - 1))/32,(wb_height+(32 - 1))/32,1);
    dim3 threadsPerBlock(32,32,1);

	cuda_check_error(hipEventRecord(start));
	row_reduction <<< dim3(num_cells_x,height,1), c, c * sizeof(rgb) >>> (gpu_data, width, height);
	cuda_check_error(hipGetLastError());
	col_reduction <<< dim3(num_cells_y,width,1), c, c * sizeof(rgb) >>> (gpu_data, width, height, c);
	cuda_check_error(hipGetLastError());
	scatter<<<blocksPerGrid, threadsPerBlock>>>(gpu_data, width, height, wb_width, wb_height, c);
	cuda_check_error(hipGetLastError());
	cuda_check_error(hipEventRecord(stop));
	cuda_check_error(hipEventSynchronize(stop));

	//TODO: global avg

    cuda_check_error(hipMemcpy(data, gpu_data, width*height*sizeof(rgb), hipMemcpyDeviceToHost));
    
    cuda_check_error(hipFree(gpu_data));

	// Output the average colour value for the image
	printf("CUDA Average image colour red = %u, green = %u, blue = %u \n",(unsigned char)global_avg.r,(unsigned char)global_avg.g,(unsigned char)global_avg.b);

	//end timing here
	double end = omp_get_wtime();
	double seconds = (end - begin);

	float cudaMs;

	hipEventElapsedTime(&cudaMs,start,stop);

	double s;
	double ms = modf(seconds,&s)*1000.0;
	printf("CUDA mode execution time took %d s and %dms (%f ms as measured by cuda)\n",(int)s,(int)ms,cudaMs);
}